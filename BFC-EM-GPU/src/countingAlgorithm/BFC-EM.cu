#include "hip/hip_runtime.h"
#include"BFC-EM.h"
#include"gpu_helper.h"
#include"timer.h"
#include <hipcub/hipcub.hpp>
#include <bits/stdc++.h>
#include <thread>
#define fileLL(a) (1LL * a / 8)
#define fileInBytes(a) (1LL * a * 8)
using namespace hipcub;
typedef long long LL;
typedef unsigned long long ULL;


__device__ __managed__ long long vertexCount;

__device__ unsigned long long hashing(long long v, long long w){
    return v * vertexCount + w;
}


__global__ void bfcWrite(int beginU, int size, long long beginIndex, long long * beginPos, int* edgeList, unsigned long long* countSize, unsigned long long * hashTable){
    int idx = blockIdx.x;
    int u = beginU + blockIdx.x;
    int vLen = beginPos[blockIdx.x + 1] - beginPos[blockIdx.x];
    for(int j = threadIdx.x; j < vLen; j += blockDim.x){
        int v = edgeList[beginPos[idx] + j - beginIndex];
        if (v >= u) break;
        unsigned long long hashBeginPos = atomicAdd(countSize, vLen - j - 1);
        for(int k = 1; k <= vLen - j - 1; k++){
            int w = edgeList[beginPos[idx] + j + k - beginIndex];
            hashTable[hashBeginPos + k - 1] = hashing(v, w);
        }
    }
}

struct node{
    unsigned long long* file;
    long long n;
    node(){}
    node(unsigned long long * _file, long long _n){
        file = _file;
        n = _n;
    }
};
const long long oneG = 1024*1024*512;

void sortPhase(int sta, int threadNum, vector<node>& hashFile){
    timer tt1;
    tt1.start();
    double ts = 0;
    long long* d_keys_in;
    long long* d_keys_out;
    GEC(hipMallocManaged(&d_keys_in, oneG));
    GEC(hipMallocManaged(&d_keys_out, oneG));
    for(long long i = sta; i < hashFile.size(); i += threadNum){
        long long num_items = hashFile[i].n;
        timer tt2;
        tt2.start();
        memcpy(d_keys_in, hashFile[i].file, fileInBytes(num_items));
        // for(int j = 0; j < 10; j++){
        //     printf("%lld %lld\n", d_keys_in[j], hashFile[i].file[j]);
        // }
        tt2.fin();
        ts += tt2.getTime();
        void     *d_temp_storage = NULL;
        size_t   temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, num_items);
        hipDeviceSynchronize();
        tt2.start();
        memcpy(hashFile[i].file, d_keys_out, fileInBytes(num_items));
        tt2.fin();
        ts += tt2.getTime();
        hipFree(d_temp_storage);
        //printf("sort file %d done!\n", i);
        fflush(stdout);
    }
    hipFree(d_keys_in);
    hipFree(d_keys_out);
    tt1.fin();
    //printf("sub sort time is %f\n", tt1.getTime());
    //printf("sub transfer time is %f\n", ts);
    fflush(stdout);
}

void sampleCount(node fileNode, vector<long long>& sumL, vector<long long>& sampleList){
    sumL.resize(sampleList.size() + 1);
    for(int i = 0; i < sumL.size(); i++) sumL[i] = 0;
    for(long long i = 0; i < fileNode.n; i++){
        sumL[lower_bound(sampleList.begin(), sampleList.end(), fileNode.file[i]) - sampleList.begin()]++;
    }
}

void alltoallGather(node fileNode, vector<long long>& prefixIndex, vector<node>& sortFileList, vector<long long>& sampleList){
    for(long long i = 0; i < fileNode.n; i++){
        int fileIndex = lower_bound(sampleList.begin(), sampleList.end(), fileNode.file[i]) - sampleList.begin();
        sortFileList[fileIndex].file[prefixIndex[fileIndex]++] = fileNode.file[i];
    }
}

void reduceCount(node fileNode, long long& ans){
    long long s = 0;
    unsigned long long pre = 0;
    ans = 0;
    for(long long i = 0; i < fileNode.n; i++){
        unsigned long long key = fileNode.file[i];
        if (key != pre){
            pre = key;
            ans += s * (s - 1) / 2;
            s = 1;
        }else s++;
    }
    ans += s * (s - 1) / 2;
}

void test(graph& g) {
    timer tall;
    tall.start();
    vertexCount = g.vertexCount;
    vector<node> hashFile;
    long long uStart = 0;
    long long check = 0;
    unsigned long long* tmpHashTable, *GTmpHashTable;
    timer t;
    t.start();
    float tCalc = 0;
    float tPre = 0;
    printf("========generate process by CPU & GPU========\n");
    while(uStart < g.vertexCount){
        long long uEnd;
        long long fileSize = 0;
        int u;
        long long uSize = 0;
        timer tt1;
        tt1.start();
        for(u = uStart; u < g.vertexCount; u++){
            uSize = 0;
            for(long long i = g.beginPos[u]; i < g.beginPos[u + 1]; i++){
                int v = g.edgeList[i];
                if (v >= u) break;
                fileSize += g.beginPos[u + 1] - i - 1;
                uSize += g.beginPos[u + 1] - i - 1;
            }
            if (fileSize >= fileLL(oneG)) break;
        }
        if (fileSize <= oneG / 8){
            uEnd = u;
        }else{
            uEnd = u - 1;
            fileSize -= uSize;
        }
        if (uEnd == g.vertexCount) uEnd -= 1;
        tt1.fin();
        tPre += tt1.getTime();
        int size = uEnd - uStart + 1;
        unsigned long long* countSize;
        hipMallocManaged(&countSize, 8);
        long long* GBeginPos;
        int* GEdgeList;
        GEC(hipMallocManaged(&GTmpHashTable, fileSize * 8));
        GEC(hipMallocManaged(&GBeginPos, (uEnd - uStart + 2) * 8));
        //printf("%d\n", (g.beginPos1[uEnd + 1] - g.beginPos1[uStart]) / 1024 / 1024 * 4);
        GEC(hipMallocManaged(&GEdgeList, (g.beginPos1[uEnd + 1] - g.beginPos1[uStart]) * 4));
        memcpy(GBeginPos, g.beginPos1 + uStart, (uEnd - uStart + 2) * 8);
        memcpy(GEdgeList, g.edgeList + g.beginPos1[uStart], (g.beginPos1[uEnd + 1] - g.beginPos1[uStart]) * 4);
        *countSize = 0;
        timer tt;
        tt.start();
        bfcWrite <<< size, 512 >>> (uStart, size, g.beginPos1[uStart], GBeginPos, GEdgeList, countSize, GTmpHashTable);
        hipDeviceSynchronize();
        tt.fin();
        check += fileSize;
        tmpHashTable = new unsigned long long[fileSize];
        hipMemcpy(tmpHashTable, GTmpHashTable, fileInBytes(fileSize), hipMemcpyDeviceToHost);
        hashFile.push_back(node(tmpHashTable, fileSize));
        tCalc += tt.getTime();
        //printf("%d %d\n", fileSize, *countSize);
        hipFree(countSize);
        hipFree(GBeginPos);
        hipFree(GEdgeList);
        hipFree(GTmpHashTable);
        uStart = uEnd + 1;
    }
    t.fin();
    printf("total time is %f\npre time is %f\ncaculate time is %f\ntransfer time is %f\n", t.getTime(), tPre, tCalc, t.getTime() - tCalc - tPre);
    printf("total count pair without deduping is %lld\n", check);
    
    fflush(stdout);
    printf("==============simple sample process by CPU===========\n");
    timer tt3;
    tt3.start();
    vector<long long> a;
    long long ttPairCheck = 0;
    for(int i = 0; i < hashFile.size(); i++){
        ttPairCheck += hashFile[i].n;
        long long step = hashFile[i].n / 8192;
        for(int j = 0; j < hashFile[i].n; j += step){
            a.push_back(hashFile[i].file[j]);
        }
    }
    sort(a.begin(), a.end());
    int sampleFileSize = hashFile.size() * 1.1;//attention *>1 to avoid greater than oneG
    int step = a.size() / sampleFileSize;
    int k = sampleFileSize - 1, i = step;
    vector<long long> sampleList;
    while(k){
        sampleList.push_back(a[i]);
        i += step;
        k--;
    }
    vector<vector<long long>> sumList;
    sumList.resize(hashFile.size());
    thread threadsS[hashFile.size()];
    for(int i = 0; i < hashFile.size(); i++){
        threadsS[i] = thread(sampleCount, hashFile[i], ref(sumList[i]), ref(sampleList));
    }
    for(int i = 0; i < hashFile.size(); i++){
        threadsS[i].join();
    }
    vector<long long> sampleFileSizeList;
    sampleFileSizeList.resize(sampleFileSize);
    for(int i = 0; i < sumList.size(); i++){
        for(int j = 0; j < sampleFileSize; j++){
            sampleFileSizeList[j] += sumList[i][j];
        }
    }
    long long ttPair = 0;
    long long mx = 0, mi = 0xfffffff;
    for(int i = 0; i < sampleFileSizeList.size(); i++){
        ttPair += sampleFileSizeList[i];
        mx = max(mx, sampleFileSizeList[i]);
        mi = min(mi, sampleFileSizeList[i]);
    }
    tt3.fin();            
    printf("sample result: %lld %lld %d\n", mx, mi, sampleFileSize);
    printf("total pair is %lld\n", ttPair);                                                                                                                       
    printf("cost time is %f\n", tt3.getTime());
    printf("===========all-to-all gather process by CPU==========\n");
    timer tt4;
    tt4.start();
    vector<vector<long long>> prefixSizeList;
    prefixSizeList.assign(hashFile.size(), vector<long long>(sampleFileSize, 0));
    for(int i = 1; i < hashFile.size(); i++){
        for(int j = 0; j < sampleFileSize; j++){
            prefixSizeList[i][j] += prefixSizeList[i - 1][j] + sumList[i - 1][j];
        }
    }
    vector<node> sortFileList;
    for(int i = 0; i < sampleFileSizeList.size(); i++){
        unsigned long long * tmpFile = new unsigned long long[sampleFileSizeList[i]];
        sortFileList.push_back(node(tmpFile, sampleFileSizeList[i]));
    } 
    thread threadsS1[hashFile.size()];
    for(int i = 0; i < hashFile.size(); i++){
        threadsS1[i] = thread(alltoallGather, hashFile[i], ref(prefixSizeList[i]), ref(sortFileList), ref(sampleList));
    }
    for(int i = 0; i < hashFile.size(); i++){
        threadsS1[i].join();
    }
    vector<vector<long long>> checkPrefixSizeList;
    checkPrefixSizeList.assign(hashFile.size(), vector<long long>(sampleFileSize, 0));
    for(int i = 0; i < hashFile.size(); i++){
        for(int j = 0; j < sampleFileSize; j++){
            if (i == 0) checkPrefixSizeList[i][j] = sumList[i][j];
            else checkPrefixSizeList[i][j] += checkPrefixSizeList[i - 1][j] + sumList[i][j];
        }
    }
    for(int i = 0; i < hashFile.size(); i++){
        for(int j = 0; j < sampleFileSize; j++){
            assert(prefixSizeList[i][j] == checkPrefixSizeList[i][j]);
        }
    }
    //     for(int i = 0; i < 10; i++){
    //     printf("%lld\n", sortFileList[0].file[i]);
    // }
    tt4.fin();
    printf("cost time is %f\n", tt4.getTime());
    fflush(stdout);
    printf("===========part sort process by GPU===========\n");
    timer t2;
    t2.start();
    int n = 2;
    thread threads2[n];
    for(int i = 0; i < n; i++){
        threads2[i] = thread(sortPhase, i, n, ref(sortFileList));
    }
    for(int i = 0; i < n; i++){
        threads2[i].join();
    }
    // for(int i = 0; i < 10; i++){
    //     printf("%lld\n", sortFileList[0].file[i]);
    // }
    t2.fin();
    printf("cost time is %f\n", t2.getTime());
    printf("==========reduce process by CPU=======\n");
    timer t3;
    t3.start();
    vector<long long> ansList(sortFileList.size());
    thread threads3[sortFileList.size()];
    for(int i = 0; i < sortFileList.size(); i++){
        threads3[i] = thread(reduceCount, sortFileList[i], ref(ansList[i]));
    }
    for(int i = 0; i < sortFileList.size(); i++){
        threads3[i].join();
    } 
    long long ans = 0;
    for(int i = 0; i < sortFileList.size(); i++){
        //printf("%lld\n", ansList[i]);
        ans += ansList[i];
    }
    t3.fin();
    printf("cost time is %f\n", t3.getTime());
    printf("=============Done=============\n");
    tall.fin();
    printf("total cost time is %f\n", tall.getTime());
    printf("ans is %lld\n", ans);
}